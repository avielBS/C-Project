#include "hip/hip_runtime.h"
#include "app.h"

Point *p = 0;
double *w = 0;
double *r = 0;


__global__ void f(double* weights, Point* points, double* result, int size, int weightSize)
{

	int i = blockIdx.x * 1000 + threadIdx.x;

	if (i < size)
	{

		int j;
		double value = 0;

		for (j = 0; j < weightSize; j++) {
			value += (weights[j] * points[i].coordinates[j]);
		}
		result[i] = value;
	}
}

hipError_t calculateWithCuda(double* weights,double* results , int size ,int weightSize )
{
	
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	////alocate the weights arrays in GPU
	//cudaStatus = hipMalloc((double**)&dev_a, weightSize * sizeof(double));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//}

	////alocate the points array in GPU
	//cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(Point));

	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//}

	////alocate the result array in GPU
	//cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(double));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//}



	// Copy input weights vector from host memory to GPU buffer.
	cudaStatus = hipMemcpy(w, weights, weightSize * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	//// Copy points vector from host memory to GPU buffer.
	//cudaStatus = hipMemcpy(dev_b, points, size * sizeof(Point), hipMemcpyHostToDevice);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy failed!");
	//}

	// Launch a kernel on the GPU with one thread for each element.
	f <<<size/1000, 1000 >>>( w, p, r,size,weightSize);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "f launch failed: %s\n", hipGetErrorString(cudaStatus));
	}


	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(results, r, size * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	return cudaStatus;
}

hipError_t setArraysInGPU(Point* points,double* weights  ,int n,int k)
{


	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	//alocate the weights arrays in GPU
	cudaStatus = hipMalloc((double**)&w, (k+1) * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	//alocate the points array in GPU
	cudaStatus = hipMalloc((void**)&p, n * sizeof(Point));

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	//alocate the result array in GPU
	cudaStatus = hipMalloc((void**)&r, n * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}



	// Copy input weights vector from host memory to GPU buffer.
	cudaStatus = hipMemcpy(w, weights, (k+1) * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	// Copy points vector from host memory to GPU buffer.
	cudaStatus = hipMemcpy(p, points, n * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	

	return cudaStatus;
}

void freeArraysInGPU()
{
	hipFree(p);
	hipFree(w);
	hipFree(r);
}